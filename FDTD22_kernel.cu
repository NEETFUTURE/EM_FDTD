//--------------------------
// FDTD22
//--------------------------


#include <hip/hip_runtime.h>
#include <math.h>
#include <omp.h>

__global__ void FDTD22(
    float *Ex, float *Ey, float *Ez,
    float *Hx, float *Hy, float *Hz,
    float *CEx, float *CEy, float *CEz,
    float *CEx_dxyz_A, float *CEy_dxyz_A, float *CEz_dxyz_A,
    float *CH_dxyz_A,
    unsigned char *val,
    unsigned long Nx, unsigned long Ny, unsigned long Nz, unsigned long Nxy,
    double Ds, double dt
    )
{
  unsigned long X, Y, Z, ID;

  X = threadIdx.x + blockIdx.x * blockDim.x;
  Y = threadIdx.y + blockIdx.y * blockDim.y;
  Z = threadIdx.z + blockIdx.z * blockDim.z;
  ID = X + Y * Nx + Z * Nx * Ny;

  //--------------------------
  //解析空間の磁界計算
  //--------------------------

  // Magnetic Field Hx //
  if(          X < Nx     &&
      Y > 0 && Y < Ny - 2 &&
      Z > 0 && Z < Nz - 2)
  {
    Hx[ID] = Hx[ID]
      - CH_dxyz_A[val[ID]] * (Ez[ID+Nx] - Ez[ID])
      + CH_dxyz_A[val[ID]] * (Ey[ID+Nxy] - Ey[ID]);
  }
  // Magnetic Field Hy //
  if(X > 0 && X<Nx-2 &&
      Y<Ny   &&
      Z > 0 && Z<Nz-2)
  {
    Hy[ID] = Hy[ID]
      - CH_dxyz_A[val[ID]] * (Ex[ID+Nxy] - Ex[ID])
      + CH_dxyz_A[val[ID]] * (Ez[ID+1] - Ez[ID]);
  }
  // Magnetic Field Hz //
  if(X > 0 && X<Nx-2 &&
      Y > 0 && Y<Ny-2 &&
      Z<Nz)
  {
    Hz[ID] = Hz[ID]
      - CH_dxyz_A[val[ID]] * (Ey[ID+1] - Ey[ID])
      + CH_dxyz_A[val[ID]] * (Ex[ID+Nx] - Ex[ID]);
  }


  // Electric Field Ex //
  if(        X<Nx   &&
      Y>=2 && Y<Ny-2 &&
      Z>=2 && Z<Nz-2)
  {
    Ex[ID] = CEx[val[ID]] * Ex[ID]
      + CEx_dxyz_A[val[ID]] * (Hz[ID]   - Hz[ID-Nx])
      - CEx_dxyz_A[val[ID]] * (Hy[ID]   - Hy[ID-Nxy]);
  }
  // Electric Field Ey //
  if(X>=2 && X<Nx-2 &&
      Y<Ny   &&
      Z>=2 && Z<Nz-2)
  {
    Ey[ID] = CEy[val[ID]] * Ey[ID]
      + CEy_dxyz_A[val[ID]] * (Hx[ID]   - Hx[ID-Nxy])
      - CEy_dxyz_A[val[ID]] * (Hz[ID]   - Hz[ID-1]);
  }
  // Electric Field Ez //
  if(X>=2 && X<Nx-2 &&
      Y>=2 && Y<Ny-2 &&
      Z<Nz  )
  {
    Ez[ID] = CEz[val[ID]] * Ez[ID]
      + CEz_dxyz_A[val[ID]] * (Hy[ID]   - Hy[ID-1])
      - CEz_dxyz_A[val[ID]] * (Hx[ID]   - Hx[ID-Nx]);
  }
}

